#include "hip/hip_runtime.h"
#include "GalerkinCuda.cuh"
#include "GalerkinData.h"

__global__ void GalerkinCuda::ElementInfluenceMatrix(float* infMatr, float* beinfo, float* fright)
{
	uint beNum = blockIdx.x / 3; // each be generate 3 equation
	uint funcNum = blockIdx.x % 3 + 1; // equation number (1..3) for fixed boundary element

	uint beNumLocal = blockIdx.y / 3; // be num which influent on fixed global be
	uint funcNumLocal = blockIdx.y % 3 + 1; // func num of be which influent on fixed global be

	uint termNum = threadIdx.x; // numeric integral term (1..numIntDiscr)

	// TO-DO: CHECK if coeff number is correct???!!!
	uint coeffNumGlobal = blockIdx.x * gridDim.y + blockIdx.y; // global number of coefficient in full influence matrix

	uint beNumInfoK = 8; // shift multiplier = size of beinfo struct

	// global boundary element info
	float xBE = beinfo[beNum * beNumInfoK+2];
	float yBE = beinfo[beNum * beNumInfoK+3];
	float alphaBE = beinfo[beNum * beNumInfoK+6];
	float lngBE = beinfo[beNum * beNumInfoK+7];

	//local boundary element info
	float xBELoc = beinfo[beNumLocal * beNumInfoK+2];
	float yBELoc = beinfo[beNumLocal * beNumInfoK+3];
	float alphaBELoc = beinfo[beNumLocal * beNumInfoK+6];
	float lngBELoc = beinfo[beNumLocal * beNumInfoK+7];

	// info for discret integral
	float discrStep = 2 * lngBE / blockDim.x;
	float xSub = -lngBE + discrStep * termNum;
	float ySub = 0;

	float xSubTransofrmed = 0, ySubTransformed = 0;

	Transform2D(xBE,yBE,alphaBE,
				0,0,0,
				xSub,ySub,
				xSubTransofrmed,ySubTransformed);

	float increment = 0;
	float frightIncrement = 0;

	switch (funcNum) {
		case 1:
			increment = discrStep * f1(xSub, lngBE) * IG(xSubTransofrmed, ySubTransformed, xBELoc, yBELoc, lngBELoc, alphaBELoc, funcNumLocal);
			if (blockIdx.x == blockIdx.y)
				frightIncrement = discrStep * f1(xSub, lngBE) * Problem::InitCondition(xSubTransofrmed, ySubTransformed);
			break;
		case 2:
			increment = discrStep * f2(xSub, lngBE) * IG(xSubTransofrmed, ySubTransformed, xBELoc, yBELoc, lngBELoc, alphaBELoc, funcNumLocal);
			if (blockIdx.x == blockIdx.y)
				frightIncrement = discrStep * f2(xSub, lngBE) * Problem::InitCondition(xSubTransofrmed, ySubTransformed);
			break;
		case 3:
			increment = discrStep * f3(xSub, lngBE) * IG(xSubTransofrmed, ySubTransformed, xBELoc, yBELoc, lngBELoc, alphaBELoc, funcNumLocal);
			if (blockIdx.x == blockIdx.y)
				frightIncrement = discrStep * f3(xSub, lngBE) * Problem::InitCondition(xSubTransofrmed, ySubTransformed);
			break;
		default:
			break;
	}

	atomicAdd(&infMatr[coeffNumGlobal], increment);
	atomicAdd(&fright[blockIdx.x], frightIncrement);
}

using namespace GalerkinMethod;

void GalerkinCuda::CalculateInfMatrix()
{ 
	if (!initialisedData) {
		printf("\nFalse while reading input data");
		return;
	}

	ResetData();

	hipSetDevice(0);

	// try to parallel maximal effective
	// we have 3N [N - number of boundary elements] equations
	// each equation is a result of numeric integral and is a sum of p_j*k [j=1..N]
	// so we need to calculate each k

	//hipDeviceProp_t prop;
	//hipGetDeviceProperties(&prop, 0);
	//printf("Device is %s\nnumber of blocks %dx%dx%d (each %dx%dx%d) = number of threads %d\n", prop.name,
	//	prop.maxGridSize[0],
	//	prop.maxGridSize[1],
	//	prop.maxGridSize[2],
	//	prop.maxThreadsDim[0],
	//	prop.maxThreadsDim[1],
	//	prop.maxThreadsDim[2],
	//	prop.maxThreadsPerBlock);

	dim3 blockSize = dim3(numIntDiscr, 1, 1); // each cofficient is a summ of numIntDiscr terms
	dim3 gridSize = dim3(beNum*3, beNum*3, 1); // each boundary element have 3 equation which consist of (beNum * 3) coefficients

	// data pointers for a kernel
	float* cudaInfMatr;
	float* cudaBeInfo;
	float* cudaFright;
	hipMalloc((void**)&cudaInfMatr, infMatrSize * sizeof(float));
	hipMalloc((void**)&cudaBeInfo, beInfoSize * sizeof(float));
	hipMalloc((void**)&cudaFright, fRightSize * sizeof(float));
	hipMemcpy(cudaInfMatr, infMatr, infMatrSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaBeInfo, beInfo, beInfoSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaFright, fRight, fRightSize * sizeof(float), hipMemcpyHostToDevice);

	ElementInfluenceMatrix <<< gridSize, blockSize >>> (cudaInfMatr, cudaBeInfo, cudaFright);


	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n",
			hipGetErrorString(cudaerr));

	hipEvent_t syncEvent;
	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(infMatr, cudaInfMatr, infMatrSize * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(fRight, cudaFright, fRightSize * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceReset();
	printf("\nSolved success!");
}

__global__ void GalerkinCuda::NodePotential(float* nodes, float* beinfo, float* coeffs)
{
	uint nodeInd = (blockIdx.x * gridDim.y + blockIdx.y) * 3;
	float x = nodes[nodeInd + 0];
	float y = nodes[nodeInd + 1];


	// local boundary element info
	uint beNumInfoK = 8; // shift multiplier = size of beinfo struct
	uint localBE = threadIdx.x;
	float xBELoc = beinfo[localBE * beNumInfoK + 2];
	float yBELoc = beinfo[localBE * beNumInfoK + 3];
	float alphaBELoc = beinfo[localBE * beNumInfoK + 6];
	float lngBELoc = beinfo[localBE * beNumInfoK + 7];

	uint func = threadIdx.y + 1;

	uint coeffInd = threadIdx.x * 3;

	float increment = 0;

	switch (func) {
		case 1:
			increment = coeffs[coeffInd + 0] * IG(x, y, xBELoc, yBELoc, lngBELoc, alphaBELoc, 1);
			break;
		case 2:
			increment = coeffs[coeffInd + 1] * IG(x, y, xBELoc, yBELoc, lngBELoc, alphaBELoc, 2);
			break;
		case 3:
			increment = coeffs[coeffInd + 2] * IG(x, y, xBELoc, yBELoc, lngBELoc, alphaBELoc, 3);
			break;
	}

	atomicAdd(&nodes[nodeInd + 2], increment);
}

void GalerkinCuda::CalculatePotentialField()
{
	if (!initialisedData || !initialisedCoeffs) {
		printf("\nFalse while reading input data");
		return;
	}

	ResetData();

	hipSetDevice(0);

	// try to parallel maximal effective
	// we have 3N [N - number of boundary elements] equations
	// each equation is a result of numeric integral and is a sum of p_j*k [j=1..N]
	// so we need to calculate each k

	//hipDeviceProp_t prop;
	//hipGetDeviceProperties(&prop, 0);
	//printf("Device is %s\nnumber of blocks %dx%dx%d (each %dx%dx%d) = number of threads %d\n", prop.name,
	//	prop.maxGridSize[0],
	//	prop.maxGridSize[1],
	//	prop.maxGridSize[2],
	//	prop.maxThreadsDim[0],
	//	prop.maxThreadsDim[1],
	//	prop.maxThreadsDim[2],
	//	prop.maxThreadsPerBlock);

	dim3 blockSize = dim3(beNum, 3, 1); // each cofficient is a summ of numIntDiscr terms
	dim3 gridSize = dim3(fdSizeX, fdSizeY, 1); // each boundary element have 3 equation which consist of (beNum * 3) coefficients

	// data pointers for a kernel
	float* cudaPotField;
	float* cudaBeInfo;
	float* cudaCoeffs;
	hipMalloc((void**)&cudaPotField, potFieldSize * sizeof(float));
	hipMalloc((void**)&cudaBeInfo, beInfoSize * sizeof(float));
	hipMalloc((void**)&cudaCoeffs, coeffsSize * sizeof(float));
	hipMemcpy(cudaPotField, potField, potFieldSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaBeInfo, beInfo, beInfoSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaCoeffs, coeffs, coeffsSize * sizeof(float), hipMemcpyHostToDevice);

	NodePotential <<< gridSize, blockSize >>> (cudaPotField, cudaBeInfo, cudaCoeffs);


	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n",
			hipGetErrorString(cudaerr));

	hipEvent_t syncEvent;
	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(potField, cudaPotField, potFieldSize * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceReset();
	printf("\nSolved success!");
}