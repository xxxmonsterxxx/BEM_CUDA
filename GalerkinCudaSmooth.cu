#include "hip/hip_runtime.h"
#include "GalerkinCudaSmooth.cuh"

#include "GalerkinData.h"

__global__ void GalerkinCudaSmooth::ElementInfluenceMatrix(float* infMatr, float* beinfo, float* fright)
{
	uint beNum = blockIdx.x; // each be generate equation
	uint beNumLocal = blockIdx.y; // be num which influent on fixed global be
	uint termNum = threadIdx.x; // numeric integral term (1..numIntDiscr)

	uint coeffNumGlobal = blockIdx.x * gridDim.y + blockIdx.y; // global number of coefficient in full influence matrix

	uint beNumInfoK1 = 19; // shift multiplier = size of beinfo struct odd
	uint beNumInfoK2 = 7;  // shift multiplier = size of beinfo struct even
	uint beNumId = (beNum % 2) ? ((beNum-1)/2) * (beNumInfoK1 + beNumInfoK2) + beNumInfoK1 : (beNum/2) * (beNumInfoK1 + beNumInfoK2);
	uint beNumLocId = (beNumLocal % 2) ? ((beNumLocal-1)/2) * (beNumInfoK1 + beNumInfoK2) + beNumInfoK1 : (beNumLocal/2) * (beNumInfoK1 + beNumInfoK2);

	//local boundary element info
	float xBELocL, yBELocL, lngBELocL, alphaBELocL; // local be info for LEFT side
	float xBELoc, yBELoc, lngBELoc, alphaBELoc;		// local be info for CENTER
	float xBELocR, yBELocR, lngBELocR, alphaBELocR; // local be info for RIGHT side
	uint BELocType = beinfo[beNumLocId + 2];

	if (BELocType == 1) {
		xBELoc = beinfo[beNumLocId + 3];
		yBELoc = beinfo[beNumLocId + 4];
		lngBELoc = beinfo[beNumLocId + 5];
		alphaBELoc = beinfo[beNumLocId + 6];
	}
	else if (BELocType == 2) {
		xBELocL = beinfo[beNumLocId + 5];
		yBELocL = beinfo[beNumLocId + 6];
		alphaBELocL = beinfo[beNumLocId + 9];
		lngBELocL = beinfo[beNumLocId + 10];

		xBELocR = beinfo[beNumLocId + 13];
		yBELocR = beinfo[beNumLocId + 14];
		alphaBELocR = beinfo[beNumLocId + 17];
		lngBELocR = beinfo[beNumLocId + 18];
	}

	// global boundary element info
	float xBE, yBE, lngBE, alphaBE;	// global be info
	uint  BEType = beinfo[beNumId + 2];

	// info for discret integral
	bool side = (termNum < (blockDim.x / 2));
	float discrStep;
	float xSub, ySub = 0;

	if (BEType == 1) {
		xBE = beinfo[beNumId + 0];
		yBE = beinfo[beNumId + 1];
		lngBE = beinfo[beNumId + 5];
		alphaBE = beinfo[beNumId + 6];

		discrStep = 2 * lngBE / blockDim.x;

		xSub = -lngBE + termNum * discrStep;
	}
	else if (BEType == 2) {
		if (side) { // left semilength
			xBE = beinfo[beNumId + 5];
			yBE = beinfo[beNumId + 6];
			alphaBE = beinfo[beNumId + 9];
			lngBE = beinfo[beNumId + 10];
			discrStep = lngBE / (blockDim.x / 2);

			xSub =  termNum * discrStep;
		}
		else { // right
			xBE = beinfo[beNumId + 13];
			yBE = beinfo[beNumId + 14];
			alphaBE = beinfo[beNumId + 17];
			lngBE = beinfo[beNumId + 18];
			discrStep = lngBE / (blockDim.x / 2);

			xSub = -lngBE + (termNum - blockDim.x/2) * discrStep;
		}
	}

	float xSubTransofrmed = 0, ySubTransformed = 0;

	Transform2D(xBE, yBE, alphaBE,
		0, 0, 0,
		xSub, ySub,
		xSubTransofrmed, ySubTransformed);

	float increment = 0;
	float frightIncrement = 0;
	float localInf;

	if (BELocType == 1)
		localInf = IG(xSubTransofrmed, ySubTransformed, xBELoc, yBELoc, lngBELoc, alphaBELoc, 2);
	else if (BELocType == 2)
		localInf = (IG(xSubTransofrmed, ySubTransformed, xBELocL, yBELocL, lngBELocL, alphaBELocL, 3) +
				    IG(xSubTransofrmed, ySubTransformed, xBELocR, yBELocR, lngBELocR, alphaBELocR, 1)   );

	if (BEType == 1) {
		increment = discrStep * f2(xSub, lngBE) * localInf;

		if (blockIdx.x == blockIdx.y)
			frightIncrement = discrStep * f2(xSub, lngBE) * Problem::InitCondition(xSubTransofrmed, ySubTransformed);
	}
	else if (BEType == 2) {
		if (side) {
			increment = discrStep * f3(xSub, lngBE) * localInf;
			if (blockIdx.x == blockIdx.y)
				frightIncrement = discrStep * f3(xSub, lngBE) * Problem::InitCondition(xSubTransofrmed, ySubTransformed);
		}
		else {
			increment = discrStep * f1(xSub, lngBE) * localInf;
			if (blockIdx.x == blockIdx.y)
				frightIncrement = discrStep * f1(xSub, lngBE) * Problem::InitCondition(xSubTransofrmed, ySubTransformed);
		}
	}

	atomicAdd(&infMatr[coeffNumGlobal], increment);
	atomicAdd(&fright[blockIdx.x], frightIncrement);
}

using namespace GalerkinMethod;

void GalerkinCudaSmooth::CalculateInfMatrix()
{
	if (!initialisedSmoothData) {
		printf("\nFalse while reading input data");
		return;
	}

	ResetData();

	hipSetDevice(0);

	// try to parallel maximal effective
	// we have 3N [N - number of boundary elements] equations
	// each equation is a result of numeric integral and is a sum of p_j*k [j=1..N]
	// so we need to calculate each k

	//hipDeviceProp_t prop;
	//hipGetDeviceProperties(&prop, 0);
	//printf("Device is %s\nnumber of blocks %dx%dx%d (each %dx%dx%d) = number of threads %d\n", prop.name,
	//	prop.maxGridSize[0],
	//	prop.maxGridSize[1],
	//	prop.maxGridSize[2],
	//	prop.maxThreadsDim[0],
	//	prop.maxThreadsDim[1],
	//	prop.maxThreadsDim[2],
	//	prop.maxThreadsPerBlock);

	dim3 blockSize = dim3(numIntDiscr, 1, 1); // each cofficient is a summ of numIntDiscr terms
	dim3 gridSize = dim3(beNum, beNum, 1); // each boundary element generate a equation which consist of beNum coefficients

	// data pointers for a kernel
	float* cudaInfMatr;
	float* cudaBeInfo;
	float* cudaFright;
	hipMalloc((void**)&cudaInfMatr, infMatrSize * sizeof(float));
	hipMalloc((void**)&cudaBeInfo, beInfoSize * sizeof(float));
	hipMalloc((void**)&cudaFright, beNum * sizeof(float));
	hipMemcpy(cudaInfMatr, infMatr, infMatrSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaBeInfo, beInfo, beInfoSize * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(cudaFright, fRight, beNum * sizeof(float), hipMemcpyHostToDevice);

	ElementInfluenceMatrix <<< gridSize, blockSize >>> (cudaInfMatr, cudaBeInfo, cudaFright);


	hipError_t cudaerr = hipDeviceSynchronize();
	if (cudaerr != hipSuccess)
		printf("kernel launch failed with error \"%s\".\n",
			hipGetErrorString(cudaerr));

	hipEvent_t syncEvent;
	hipEventCreate(&syncEvent);
	hipEventRecord(syncEvent, 0);
	hipEventSynchronize(syncEvent);

	hipMemcpy(infMatr, cudaInfMatr, infMatrSize * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(fRight, cudaFright, beNum * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceReset();
	printf("\nSolved success!");
}