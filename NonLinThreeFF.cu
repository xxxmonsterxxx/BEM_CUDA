
#include <hip/hip_runtime.h>
//#define _USE_MATH_DEFINES
//
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <math.h>
//#include <stdio.h>
//#include <conio.h>
//#include <iostream>
//#include <fstream>
//#include <ctime>
//#include <string>
//
//void GetInputAndCalcInfluence();
//void GetInputAndCalcDistr();
//void CreateMatrix(double* Matrixij, double* BECoords, double* BE, int BeNumber, int BEInfoSize);
//void CreateNodes(double* Matrixij, int size, double* Coeffs, int BeNumber, double* BE, int BEInfoSize);
//
//// cuda function create influence matrix with pointer 'Matrixij' for boundary condition in points with coords 'BECoords',
//// but boundary elements are in coords 'BE'
//__global__ void MatrixCreation(double* Matrixij, double* BECoords, double* BE)
//{
//	int i = blockIdx.x;
//	int j0 = threadIdx.x;
//	int index = i * blockDim.x + j0;
//
//	// coords of calculation point which is be coords with shift to avoid undeterminated state
//	double x = BECoords[i];
//	double y = BECoords[i + 1];
//
//	if (j0 % 2)
//	{
//		int j = (j0 - 1) / 2 * (19 + 7) + 19;
//		Matrixij[index] = (-6 * atanf((-BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) / ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]))) * ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6])) *
//			(3 * powf(BE[j + 5], 2) + powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) - 3 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) +
//			(BE[j + 5] - (-BE[j + 3] + x) * cosf(BE[j + 6]) - (-BE[j + 4] + y) * sinf(BE[j + 6])) * (-16 * powf(BE[j + 5], 2) - 6 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) +
//				5 * BE[j + 5] * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) + 11 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) -
//			3 * logf(powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) + powf(-BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) *
//			(powf(BE[j + 5], 3) + 3 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) -
//				powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 3) + 3 * powf(BE[j + 5], 2) * (-BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])))) / (36. * powf(BE[j + 5], 2) * M_PI) -
//			(-6 * atanf((BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) / ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]))) * ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6])) *
//				(3 * powf(BE[j + 5], 2) + powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) - 3 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) +
//				(-BE[j + 5] - (-BE[j + 3] + x) * cosf(BE[j + 6]) - (-BE[j + 4] + y) * sinf(BE[j + 6])) * (-16 * powf(BE[j + 5], 2) - 6 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) -
//					5 * BE[j + 5] * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) + 11 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) -
//				3 * logf(powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) + powf(BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) *
//				(-powf(BE[j + 5], 3) + 3 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) -
//					powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 3) + 3 * powf(BE[j + 5], 2) * (BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])))) / (36. * powf(BE[j + 5], 2) * M_PI);
//	}
//	else
//	{
//		int j = j0 / 2 * (19 + 7);
//		Matrixij[index] = -(12 * atanf((BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) / ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]))) * ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9])) *
//			(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - 3 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//				3 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//			(BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) * (9 * BE[j + 10] * (-BE[j + 10] + 3 * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]))) +
//				2 * (2 * powf(BE[j + 10], 2) - 6 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - 5 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) +
//					11 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2))) +
//			logf(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) + powf(BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) *
//			(9 * BE[j + 10] * (powf(BE[j + 10], 2) + powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//				6 * (-powf(BE[j + 10], 3) + 3 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//					powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 3)))) / (144. * powf(BE[j + 10], 2) * M_PI) +
//			(12 * atanf((-BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) / ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]))) * ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9])) *
//				(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - 3 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//					3 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//				(-BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) * (9 * BE[j + 10] * (BE[j + 10] + 3 * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]))) +
//					2 * (2 * powf(BE[j + 10], 2) - 6 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) + 5 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) +
//						11 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2))) +
//				logf(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) + powf(-BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) *
//				(9 * BE[j + 10] * (powf(BE[j + 10], 2) + powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//					6 * (powf(BE[j + 10], 3) + 3 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//						powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 3)))) / (144. * powf(BE[j + 10], 2) * M_PI) -
//			(-12 * atanf((-BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) / ((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]))) *
//				((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17])) * (powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) +
//					3 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) - 3 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//				(-BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) * (9 * BE[j + 18] * (BE[j + 18] + 3 * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]))) -
//					2 * (2 * powf(BE[j + 18], 2) - 6 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) + 5 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						11 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2))) +
//				logf(powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) + powf(-BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) *
//				(9 * BE[j + 18] * (powf(BE[j + 18], 2) + powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) - powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//					6 * (-powf(BE[j + 18], 3) - 3 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 3)))) / (144. * powf(BE[j + 18], 2) * M_PI) +
//			(-12 * atanf((BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) / ((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]))) *
//				((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17])) * (powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) +
//					3 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) - 3 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//				(BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) * (9 * BE[j + 18] * (-BE[j + 18] + 3 * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]))) -
//					2 * (2 * powf(BE[j + 18], 2) - 6 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) - 5 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						11 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2))) +
//				logf(powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) + powf(BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) *
//				(9 * BE[j + 18] * (powf(BE[j + 18], 2) + powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) - powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//					6 * (powf(BE[j + 18], 3) - 3 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 3)))) / (144. * powf(BE[j + 18], 2) * M_PI);
//	}
//}
//
//// cuda function calculate one term for one be coord and add it to value belong to be coord
//__global__ void CalculateNodes(double* Matrixij, double* Coeff, double* BE)
//{
//	int i = blockIdx.x; // index by x axis
//	int j0 = blockIdx.y; // index by be element
//	int k = threadIdx.x; // index by y axis
//
//	int index = (i * blockDim.x + k) * 3;
//
//	double x = Matrixij[index];
//	double y = Matrixij[index + 1];
//
//	if (j0 % 2) {
//		int j = (j0 - 1) / 2 * (19 + 7) + 19;
//		double increment = Coeff[j0] * ((-6 * atanf((-BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) / ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]))) * ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6])) *
//			(3 * powf(BE[j + 5], 2) + powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) - 3 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) +
//			(BE[j + 5] - (-BE[j + 3] + x) * cosf(BE[j + 6]) - (-BE[j + 4] + y) * sinf(BE[j + 6])) * (-16 * powf(BE[j + 5], 2) - 6 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) +
//				5 * BE[j + 5] * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) + 11 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) -
//			3 * logf(powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) + powf(-BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) *
//			(powf(BE[j + 5], 3) + 3 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) -
//				powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 3) + 3 * powf(BE[j + 5], 2) * (-BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])))) / (36. * powf(BE[j + 5], 2) * M_PI) -
//			(-6 * atanf((BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) / ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]))) * ((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6])) *
//				(3 * powf(BE[j + 5], 2) + powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) - 3 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) +
//				(-BE[j + 5] - (-BE[j + 3] + x) * cosf(BE[j + 6]) - (-BE[j + 4] + y) * sinf(BE[j + 6])) * (-16 * powf(BE[j + 5], 2) - 6 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) -
//					5 * BE[j + 5] * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) + 11 * powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) -
//				3 * logf(powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) + powf(BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 2)) *
//				(-powf(BE[j + 5], 3) + 3 * powf((-BE[j + 4] + y) * cosf(BE[j + 6]) + (BE[j + 3] - x) * sinf(BE[j + 6]), 2) * ((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])) -
//					powf((-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6]), 3) + 3 * powf(BE[j + 5], 2) * (BE[j + 5] + (-BE[j + 3] + x) * cosf(BE[j + 6]) + (-BE[j + 4] + y) * sinf(BE[j + 6])))) / (36. * powf(BE[j + 5], 2) * M_PI));
//		atomicAdd(&(Matrixij[index + 2]), increment);
//	}
//	else {
//		int j = j0 / 2 * (19 + 7);
//		double increment = Coeff[j0] * (-(12 * atanf((BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) / ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]))) * ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9])) *
//			(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - 3 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//				3 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//			(BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) * (9 * BE[j + 10] * (-BE[j + 10] + 3 * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]))) +
//				2 * (2 * powf(BE[j + 10], 2) - 6 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - 5 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) +
//					11 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2))) +
//			logf(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) + powf(BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) *
//			(9 * BE[j + 10] * (powf(BE[j + 10], 2) + powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//				6 * (-powf(BE[j + 10], 3) + 3 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//					powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 3)))) / (144. * powf(BE[j + 10], 2) * M_PI) +
//			(12 * atanf((-BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) / ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]))) * ((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9])) *
//				(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - 3 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//					3 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//				(-BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) * (9 * BE[j + 10] * (BE[j + 10] + 3 * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]))) +
//					2 * (2 * powf(BE[j + 10], 2) - 6 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) + 5 * BE[j + 10] * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) +
//						11 * powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2))) +
//				logf(powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) + powf(-BE[j + 10] + (-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) *
//				(9 * BE[j + 10] * (powf(BE[j + 10], 2) + powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) - powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 2)) +
//					6 * (powf(BE[j + 10], 3) + 3 * powf((-BE[j + 6] + y) * cosf(BE[j + 9]) + (BE[j + 5] - x) * sinf(BE[j + 9]), 2) * ((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9])) -
//						powf((-BE[j + 5] + x) * cosf(BE[j + 9]) + (-BE[j + 6] + y) * sinf(BE[j + 9]), 3)))) / (144. * powf(BE[j + 10], 2) * M_PI) -
//			(-12 * atanf((-BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) / ((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]))) *
//				((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17])) * (powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) +
//					3 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) - 3 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//				(-BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) * (9 * BE[j + 18] * (BE[j + 18] + 3 * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]))) -
//					2 * (2 * powf(BE[j + 18], 2) - 6 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) + 5 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						11 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2))) +
//				logf(powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) + powf(-BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) *
//				(9 * BE[j + 18] * (powf(BE[j + 18], 2) + powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) - powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//					6 * (-powf(BE[j + 18], 3) - 3 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 3)))) / (144. * powf(BE[j + 18], 2) * M_PI) +
//			(-12 * atanf((BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) / ((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]))) *
//				((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17])) * (powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) +
//					3 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) - 3 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//				(BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) * (9 * BE[j + 18] * (-BE[j + 18] + 3 * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]))) -
//					2 * (2 * powf(BE[j + 18], 2) - 6 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) - 5 * BE[j + 18] * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						11 * powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2))) +
//				logf(powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) + powf(BE[j + 18] + (-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) *
//				(9 * BE[j + 18] * (powf(BE[j + 18], 2) + powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) - powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 2)) +
//					6 * (powf(BE[j + 18], 3) - 3 * powf((-BE[j + 14] + y) * cosf(BE[j + 17]) + (BE[j + 13] - x) * sinf(BE[j + 17]), 2) * ((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17])) +
//						powf((-BE[j + 13] + x) * cosf(BE[j + 17]) + (-BE[j + 14] + y) * sinf(BE[j + 17]), 3)))) / (144. * powf(BE[j + 18], 2) * M_PI));
//		atomicAdd(&(Matrixij[index + 2]), increment);
//	}
//}
//
//// function using CUDA function "MatrixCreation" create influence matrix by pointer Matrixij[size*size]
//void CreateMatrix(double* Matrixij, double* BECoords, double* BE, int BeNumber, int BEInfoSize)
//{
//	double* dev_a, * dev_b, * dev_c;
//
//	cudaSetDevice(0);
//
//	cudaMalloc((void**)&dev_a, BeNumber * BeNumber * sizeof(double));
//	cudaMalloc((void**)&dev_b, 2 * BeNumber * sizeof(double));
//	cudaMalloc((void**)&dev_c, BEInfoSize * sizeof(double));
//	cudaMemcpy(dev_a, Matrixij, BeNumber * BeNumber * sizeof(double), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_b, BECoords, BeNumber * 2 * sizeof(double), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_c, BE, BEInfoSize * sizeof(double), cudaMemcpyHostToDevice);
//
//	dim3 blockSize = dim3(BeNumber, 1, 1);
//	dim3 gridSize = dim3(BeNumber, 1, 1);
//
//	MatrixCreation << <gridSize, blockSize >> > (dev_a, dev_b, dev_c);
//
//	cudaEvent_t syncEvent;
//	cudaEventCreate(&syncEvent);
//	cudaEventRecord(syncEvent, 0);
//	cudaEventSynchronize(syncEvent);
//
//	cudaMemcpy(Matrixij, dev_a, BeNumber * BeNumber * sizeof(double), cudaMemcpyDeviceToHost);
//}
//
//void CreateNodes(double* Matrixij, int size, double* Coeffs, int BeNumber, double* BE, int BEInfoSize)
//{
//	double* dev_a, * dev_b, * dev_c;
//
//	cudaSetDevice(0);
//
//	cudaMalloc((void**)&dev_a, 3 * size * size * sizeof(double));
//	cudaMalloc((void**)&dev_b, BeNumber * sizeof(double));
//	cudaMalloc((void**)&dev_c, BEInfoSize * sizeof(double));
//	cudaMemcpy(dev_a, Matrixij, 3 * size * size * sizeof(double), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_b, Coeffs, BeNumber * sizeof(double), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_c, BE, BEInfoSize * sizeof(double), cudaMemcpyHostToDevice);
//
//	dim3 blockSize = dim3(size, 1, 1);
//	dim3 gridSize = dim3(size, BeNumber, 1);
//
//	CalculateNodes << < gridSize, blockSize >> > (dev_a, dev_b, dev_c);
//
//	cudaEvent_t syncEvent;
//	cudaEventCreate(&syncEvent);
//	cudaEventRecord(syncEvent, 0);
//	cudaEventSynchronize(syncEvent);
//
//	cudaMemcpy(Matrixij, dev_a, 3 * size * size * sizeof(double), cudaMemcpyDeviceToHost);
//}
//
//void GetInputAndCalcInfluence()
//{
//	double* Coords, * BE;
//	double* Matrixij;
//
//	int bediscr_array[] = { 5,10,15,20,25,30,35 };
//	string input_file_name;
//	string output_file_name;
//
//	for (int i = 0; i < 7; i++)
//	{
//		int bediscr = bediscr_array[i];
//		int CoordsNumber, BEInfoSize;
//
//		input_file_name = "D:/Docs/article_03_20/data/shifted_coords" + to_string(bediscr) + ".txt";
//		ifstream in;
//		in.open(input_file_name);
//
//		in >> CoordsNumber;
//		Coords = new double[CoordsNumber];
//
//		for (int i = 0; i < CoordsNumber; i++)
//			in >> Coords[i];
//
//		in.close();
//
//		input_file_name = "D:/Docs/article_03_20/data/beinfo" + to_string(bediscr) + ".txt";
//		in.open(input_file_name);
//
//		in >> BEInfoSize;
//		BE = new double[BEInfoSize];
//
//		for (int i = 0; i < BEInfoSize; i++)
//			in >> BE[i];
//
//		in.close();
//
//		int benumb = CoordsNumber / 2;
//
//		Matrixij = new double[benumb * benumb];
//
//		unsigned int start_time;
//		unsigned int end_time;
//		unsigned int search_time = 0;
//
//		/////////////// CUDA method //////////////////////////
//		CreateMatrix(Matrixij, Coords, BE, benumb, BEInfoSize);
//		start_time = clock();
//		CreateMatrix(Matrixij, Coords, BE, benumb, BEInfoSize);
//		end_time = clock();
//		search_time = end_time - start_time;
//
//		output_file_name = "D:/Docs/article_03_20/CUDA_output/matr" + to_string(bediscr) + ".txt";
//		ofstream out;
//		out.open(output_file_name);
//
//		for (int i = 0; i < benumb; i++)
//			for (int j = 0; j < benumb; j++)
//				out << Matrixij[i * benumb + j] << '\n';
//		out << search_time;
//		out.close();
//
//		/////////////// sequential method ////////////////////
//		start_time = clock();
//		MatrixCreationSeq(Matrixij, Coords, BE, benumb);
//		end_time = clock();
//		search_time = end_time - start_time;
//
//		output_file_name = "D:/Docs/article_03_20/SEQ_output/matr" + to_string(bediscr) + ".txt";
//		out.open(output_file_name);
//
//		for (int i = 0; i < benumb; i++)
//			for (int j = 0; j < benumb; j++)
//				out << Matrixij[i * benumb + j] << '\n';
//		out << search_time;
//		out.close();
//
//		///////////////////////////////////////////////////////
//
//		delete(Matrixij);
//		delete(BE);
//		delete(Coords);
//	}
//}
//
//void GetInputAndCalcDistr()
//{
//	int bediscr_array[] = { 5,10,15,20,25,30,35 };
//	int areadiscr_array[] = { 10, 20, 30, 40, 50 };
//	double* BE, * Coefs;
//	int CoefsNumb, BEInfoSize;
//	string input_file_name;
//	string output_file_name;
//	int bediscr, areadiscr;
//
//	for (int k = 0; k < 7; k++)
//	{
//		bediscr = bediscr_array[k];
//		input_file_name = "D:/Docs/article_03_20/data/coefs" + to_string(bediscr) + ".txt";
//		ifstream in;
//		in.open(input_file_name);
//
//		in >> CoefsNumb;
//		Coefs = new double[CoefsNumb];
//
//		for (int i = 0; i < CoefsNumb; i++)
//			in >> Coefs[i];
//
//		in.close();
//
//		input_file_name = "D:/Docs/article_03_20/data/beinfo" + to_string(bediscr) + ".txt";
//		in.open(input_file_name);
//
//		in >> BEInfoSize;
//		BE = new double[BEInfoSize];
//
//		for (int i = 0; i < BEInfoSize; i++)
//			in >> BE[i];
//
//		in.close();
//
//		for (int s = 0; s < 5; s++)
//		{
//			areadiscr = areadiscr_array[s];
//			float discrx = (WIDTH - 2 * EPS) / (areadiscr - 1);
//			float discry = (HEIGHT - 2 * EPS) / (areadiscr - 1);
//
//			int matrixsize = areadiscr * areadiscr * 3;
//			double* Matrixij = new double[matrixsize];
//
//			for (int i = 0; i < areadiscr; i++)
//				for (int j = 0; j < areadiscr; j++)
//				{
//					int idx = (i * areadiscr + j) * 3;
//					Matrixij[idx] = -WIDTH / 2 + EPS + discrx * i;
//					Matrixij[idx + 1] = -EPS - discry * j;
//					Matrixij[idx + 2] = 0;
//				}
//
//			unsigned int start_time;
//			unsigned int end_time;
//			unsigned int search_time = 0;
//
//			CreateNodes(Matrixij, areadiscr, Coefs, CoefsNumb, BE, BEInfoSize); // startup calculation to activate cuda memory
//			start_time = clock();
//			CreateNodes(Matrixij, areadiscr, Coefs, CoefsNumb, BE, BEInfoSize);
//			end_time = clock();
//			search_time = end_time - start_time;
//
//			output_file_name = "D:/Docs/article_03_20/CUDA_Output/node" + to_string(bediscr) + "-" + to_string(areadiscr) + ".txt";
//			ofstream out;
//			out.open(output_file_name);
//
//			for (int i = 0; i < matrixsize; i = i + 3)
//				out << Matrixij[i] << " " << Matrixij[i + 1] << " " << Matrixij[i + 2] << '\n';
//			out << search_time;
//
//			out.close();
//
//			/////////////// sequential method ////////////////////
//			start_time = clock();
//			CalculateNodesSeq(Matrixij, areadiscr, Coefs, CoefsNumb, BE);
//			end_time = clock();
//			search_time = end_time - start_time;
//
//			output_file_name = "D:/Docs/article_03_20/SEQ_output/node" + to_string(bediscr) + "-" + to_string(areadiscr) + ".txt";
//			out.open(output_file_name);
//
//			for (int i = 0; i < matrixsize; i = i + 3)
//				out << Matrixij[i] << " " << Matrixij[i + 1] << " " << Matrixij[i + 2] << '\n';
//			out << search_time;
//			out.close();
//
//			///////////////////////////////////////////////////////
//
//			delete(Matrixij);
//		}
//
//		delete(Coefs);
//		delete(BE);
//	}
//}