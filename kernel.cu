#include "hip/hip_runtime.h"
#include "GalerkinData.h"
#include "GalerkinCuda.cuh"
#include "GalerkinSerial.h"
#include "GalerkinCudaSmooth.cuh"
#include "GalerkinSmoothSerial.h"
#include <ctime>

using namespace GalerkinMethod;

int main()
{
    //uint start_time;
    //uint end_time;
    //uint spand_time;

    ////----------------------------------------------------
    ////----------------------------------------------------
    //GalerkinMethod::InitInputData();
    ////----------------------------------------------------
    //start_time = clock();
    //GalerkinCuda::CalculateInfMatrix();
    //end_time = clock();
    //spand_time = end_time - start_time;
    //printf("\nCuda inf matr computation time = %d\n", spand_time);

    //start_time = clock();
    //GalerkinSerial::CalculateInfMatrix();
    //end_time = clock();
    //spand_time = end_time - start_time;
    //printf("\nSeq. inf matr computation time = %d\n", spand_time);
    ////----------------------------------------------------
    //GalerkinMethod::InitInputSmoothData();
    ////----------------------------------------------------
    //start_time = clock();
    //GalerkinCudaSmooth::CalculateInfMatrix();
    //end_time = clock();
    //spand_time = end_time - start_time;
    //printf("\nCuda smooth inf matr computation time = %d\n", spand_time);

    //start_time = clock();
    //GalerkinSmoothSerial::CalculateInfMatrix();
    //end_time = clock();
    //spand_time = end_time - start_time;
    //printf("\nSeq. smooth inf matr computation time = %d\n", spand_time);
    ////----------------------------------------------------
    ////----------------------------------------------------
    //start_time = clock();
    //GalerkinCuda::CalculatePotentialField();
    //end_time = clock();
    //spand_time = end_time - start_time;
    //printf("\nCuda calc potential field computation time = %d\n", spand_time);

    //start_time = clock();
    //GalerkinSerial::CalculatePotentialField();
    //end_time = clock();
    //spand_time = end_time - start_time;
    //printf("\Seq. calc potential field computation time = %d\n", spand_time);
    //----------------------------------------------------
    //start_time = clock();
    GalerkinMethod::InitInputSmoothData();
    GalerkinSmoothSerial::CalculatePotentialField();
    for (int i = 0; i < potFieldSize; i++) {
        printf("\n%f", potField[i]);
    }

    //end_time = clock();
    //spand_time = end_time - start_time;
    //printf("\nCuda smooth calc potential field computation time = %d\n", spand_time);

    /*start_time = clock();
    GalerkinSmoothSerial::CalculatePotentialField();
    end_time = clock();
    spand_time = end_time - start_time;
    printf("\Seq. smooth calc potential field computation time = %d\n", spand_time);*/
    //----------------------------------------------------
    

    return 0;
}