#include "hip/hip_runtime.h"
#include "GalerkinData.h"
#include "GalerkinCuda.cuh"
#include "GalerkinSerial.h"
#include "GalerkinCudaSmooth.cuh"
#include "GalerkinSmoothSerial.h"
#include <ctime>

using namespace GalerkinMethod;

int main()
{
    uint start_time;
    uint end_time;
    uint spand_time;

    GalerkinMethod::InitInputData();
//    GalerkinCuda::CalculatePotentialField();
    GalerkinSerial::CalculatePotentialField();
    for (int i = 0; i < potFieldSize; i++)
        printf("\n%f", potField[i]);
    /*for (int i = 0; i < 4; i++) {
        start_time = clock();
        GalerkinCuda::CalculateInfMatrix();
        end_time = clock();
        spand_time = end_time - start_time;
        printf("\nCuda computation time = %d\n", spand_time);
    }

    start_time = clock();
    GalerkinSerial::Solve();
    end_time = clock();
    spand_time = end_time - start_time;
    printf("\nSeq. computation time = %d\n", spand_time);*/

    /*GalerkinMethod::InitInputSmoothData();
    for (int i = 0; i < 4; i++) {
        start_time = clock();
        GalerkinCudaSmooth::CalculateInfMatrix();
        end_time = clock();
        spand_time = end_time - start_time;
        printf("\nCuda computation time = %d\n", spand_time);
    }

    start_time = clock();
    GalerkinSmoothSerial::CalculateInfMatrix();
    end_time = clock();
    spand_time = end_time - start_time;
    printf("\nSeq. computation time = %d\n", spand_time);*/

    return 0;
}